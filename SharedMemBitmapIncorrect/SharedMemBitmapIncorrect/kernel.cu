#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include "book.h"
#include "cpu_bitmap.h"

#include <stdio.h>
#include <math.h>

#define DIM 1024
#define PI 3.14

__global__ void kernel(unsigned char* ptr) {
	//map from threadIdx/blockIdx to pixel position.
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	int offset = x + y * blockDim.x * gridDim.x;

	__shared__ float shared[16][16];

	//calculate the value at that position.
	const float period = 128.f;

	shared[threadIdx.x][threadIdx.y] = 255 * (sinf(x*2.f*PI/period)+1.f) * (sinf(y*2.f*PI/period)+1.f) / 4.f;

	//__syncthreads();

	ptr[offset*4 + 0] = 0;
	ptr[offset*4 + 1] = shared[15-threadIdx.x][15 - threadIdx.y];
	ptr[offset*4 + 2] = 0;
	ptr[offset*4 + 3] = 255;
}

int main(int argc, char** argv) {
	CPUBitmap bitmap(DIM, DIM);
	unsigned char* dev_bitmap;

	hipMalloc((void**)&dev_bitmap, bitmap.image_size());

	dim3 grids(DIM/16, DIM/16);
	dim3 threads(16, 16);

	kernel<<<grids, threads>>>(dev_bitmap);

	hipMemcpy(bitmap.get_ptr(), dev_bitmap, bitmap.image_size(), hipMemcpyDeviceToHost);

	bitmap.display_and_exit();

	hipFree(dev_bitmap);
}